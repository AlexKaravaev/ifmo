
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;



__global__ void similarity(const int* A, int *D, int w, int h){
    // Compute each thread's global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int temp_sum = 0;
    int diff = 0;

    if ((row < h) && (col < h)){
        for (int k = 0; k < w; k++){
            diff = (A[row*w + k] - A[col*w + k]);
            temp_sum += diff*diff;
            printf(
                "row*n+col: %d row*n+k: %d row: %d, col: %d A[%d,%d]: %d"
                " A[%d,%d]: %d sum: %d\n ", 
                row*w + k, col*h + k, row, col, row, col,A[row*w + k],
                row, col, A[col*h + k], diff);
            }
        D[row * h + col] = temp_sum;
    }
}


vector<int> similarity_serial(vector<int> A, int w, int h){
    // Compute each thread's global row and column index

    vector<int> D;
    for (int row = 0; row < h; row++){
        for (int col = 0; col < h; col++)
        {
            int temp_sum = 0;
            int diff = 0;

            for (int k = 0; k < w; k++){
          
                diff = (A[row*w + k] - A[col*w + k]);
                temp_sum += diff*diff;
                printf(
                    "row*n+col: %d row*n+k: %d row: %d, col: %d A[%d,%d]: %d"
                    " A[%d,%d]: %d sum: %d\n", 
                    row*h + k, col*w + k, row, col, row, col, A[row*h + k],
                    row,col, A[col*w + k], diff);
                }
            D.push_back(temp_sum);

        }
    }

    return D;
}
int main() {

    // Define matrix size
    int w = 3;
    int h = 5;

    // Size (in bytes) of matrix
    size_t a_bytes = w * h * sizeof(int);
    size_t d_bytes = h * h * sizeof(int);

    vector<int> A(w * h);
    vector<int> D(h * h);

    // Initialize matrices
    //generate(A.begin(), A.end(), []() { return rand() % 100; });

   A[0] = 0;
   A[1] = 1;
   A[2] = 1;
   A[3] = 4;
   A[4] = 0;
   A[5] = 2;
   A[6] = 3;
   A[7] = 1;
   A[8] = 1;
   A[9] = 0;
   A[10] = 0;
   A[11] = 0;
   A[12] = 2;
   A[13] = 1;
   A[14] = 2;

    for (int i=0;i<h;i++){
        for (int j=0;j<w;j++)
            std::cout << A[i*w + j] << " ";
        std::cout << "\n";
    }
     std::cout << "\n";
      // Allocate device memory
      int *d_a, *d_d;
      hipMalloc(&d_a, a_bytes);
      hipMalloc(&d_d, d_bytes);

      // Copy data to the device

      hipMemcpy(d_a, A.data(), a_bytes, hipMemcpyHostToDevice);
      hipMemcpy(d_d, D.data(), d_bytes, hipMemcpyHostToDevice);

      // Threads per CTA dimension
      int THREADS = h;

      // Blocks per grid dimension (assumes THREADS divides N evenly)
      int BLOCKS = h * h / THREADS;

      // Use dim3 structs for block  and grid dimensions
      dim3 threads(THREADS, THREADS);
      dim3 blocks(BLOCKS, BLOCKS);

      // Launch kernel
      similarity<<<blocks, threads>>>(d_a, d_d, w, h);

  // Copy back to the host
  hipMemcpy(D.data(), d_d, d_bytes, hipMemcpyDeviceToHost);

  // Check result
  // verify_result(h_a, h_b, h_c, N);

  cout << "COMPLETED SUCCESSFULLY\n";
  for (int i=0;i<h;i++){
      for (int j=0;j<h;j++)
      std::cout << D[i*h + j] << " ";
      std::cout << "\n";
    }

  std::cout << "\n";
  auto serial = similarity_serial(A,w,h);
  for (int i=0;i<h;i++){
      for (int j=0;j<h;j++)
      std::cout << serial[i*h + j] << " ";
      std::cout << "\n";
    }

  // Free memory on device
  hipFree(d_a);
  hipFree(d_d);


  return 0;
}