
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <chrono>
//#define TEST
#define BLOCKSIZE 16

using std::cout;
using std::generate;
using std::vector;
using namespace std::chrono;

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__global__ void similarity(const int *A, int *D, int w, int h)
{
    // Compute each thread's global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int temp_sum = 0;
    int diff = 0;

    if ((row < h) && (col < h))
    {
        for (int k = 0; k < w; k++)
        {
            diff = (A[row * w + k] - A[col * w + k]);
            temp_sum += diff * diff;
        }
        D[row * h + col] = temp_sum;
    }
}


__global__ void similarity_shared(const int *A, int *D, int w, int h)
{
    // Compute each thread's global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x, ty = threadIdx.y;

    int temp_sum = 0;
    int diff = 0;

    __shared__ float as[BLOCKSIZE][BLOCKSIZE];
    __shared__ float as_pow[BLOCKSIZE][BLOCKSIZE];

    if ((row < h) && (col < h))
    {
        for (int k = 0; k < w; k++)
        {
            as[tx][ty] = (A[row * w + k] - A[col * w + k]);
            __syncthreads();
            as_pow[tx][ty] = as[tx][ty]*as[tx][ty];
            __syncthreads();
            temp_sum += as_pow[tx][ty];
            __syncthreads();
        }
        D[row * h + col] = temp_sum;
    }
}

vector<int> similarity_serial(vector<int> A, int w, int h)
{
    vector<int> D;
    for (int row = 0; row < h; row++)
    {
        for (int col = 0; col < h; col++)
        {
            int temp_sum = 0;
            int diff = 0;

            for (int k = 0; k < w; k++)
            {

                diff = (A[row * w + k] - A[col * w + k]);
                temp_sum += diff * diff;
            }
            D.push_back(temp_sum);
        }
    }

    return D;
}

int main()
{

    int w = 32 * 20;
    int h = 32 * 30;

    size_t a_bytes;
    size_t d_bytes;
#ifdef TEST

    w = 3;
    h = 5;
    a_bytes = w * h * sizeof(int);
    d_bytes = h * h * sizeof(int);

#endif
    a_bytes = w * h * sizeof(int);
    d_bytes = h * h * sizeof(int);

    vector<int> A(w * h);
#ifdef TEST

    A[0] = 0;
    A[1] = 1;
    A[2] = 1;
    A[3] = 4;
    A[4] = 0;
    A[5] = 2;
    A[6] = 3;
    A[7] = 1;
    A[8] = 1;
    A[9] = 0;
    A[10] = 0;
    A[11] = 0;
    A[12] = 2;
    A[13] = 1;
    A[14] = 2;
    for (int i = 0; i < h; i++)
    {
        for (int j = 0; j < w; j++)
            std::cout << A[i * w + j] << " ";
        std::cout << "\n";
    }
    std::cout << "\n";
#else
    generate(A.begin(), A.end(), []() { return rand() % 100; });

#endif
    vector<int> D(h * h);

    int *d_a, *d_d;
    gpuErrchk(hipMalloc(&d_a, a_bytes));
    gpuErrchk(hipMalloc(&d_d, d_bytes));

    gpuErrchk(hipMemcpy(d_a, A.data(), a_bytes, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_d, D.data(), d_bytes, hipMemcpyHostToDevice));

#ifdef TEST
    int THREADS = w;
    int BLOCKS  = h * w / THREADS;
#else
    int THREADS = 16;
    int BLOCKS = BLOCKSIZE;
#endif

    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    auto start = high_resolution_clock::now();
    similarity<<<blocks, threads>>>(d_a, d_d, w, h);
    auto stop = high_resolution_clock::now();

    auto duration = duration_cast<nanoseconds>(stop - start);

    gpuErrchk(hipMemcpy(D.data(), d_d, d_bytes, hipMemcpyDeviceToHost));

    cout << "CUDA COMPLETED SUCCESSFULLY in " << duration.count() << " nanoseconds \n";
#ifdef TEST
    for (int i = 0; i < h; i++)
    {
        for (int j = 0; j < h; j++)
            std::cout << D[i * h + j] << " ";
        std::cout << "\n";
    }

    std::cout << "\n";
#endif

    start = high_resolution_clock::now();
    similarity_shared<<<blocks, threads>>>(d_a, d_d, w, h);
    stop = high_resolution_clock::now();

    duration = duration_cast<nanoseconds>(stop - start);
    
    gpuErrchk(hipMemcpy(D.data(), d_d, d_bytes, hipMemcpyDeviceToHost));

    cout << "CUDA SHARED COMPLETED SUCCESSFULLY in " << duration.count() << " nanoseconds \n";
#ifdef TEST
    for (int i = 0; i < h; i++)
    {
        for (int j = 0; j < h; j++)
            std::cout << D[i * h + j] << " ";
        std::cout << "\n";
    }

    std::cout << "\n";
#endif


    start = high_resolution_clock::now();
    auto serial = similarity_serial(A, w, h);
    stop = high_resolution_clock::now();
    duration = duration_cast<nanoseconds>(stop - start);
    cout << "SERIAL COMPLETED SUCCESSFULLY in " << duration.count() << " nanoseconds \n";

#ifdef TEST
    for (int i = 0; i < h; i++)
    {
        for (int j = 0; j < h; j++)
            std::cout << serial[i * h + j] << " ";
        std::cout << "\n";
    }
#endif
    gpuErrchk(hipFree(d_a));
    gpuErrchk(hipFree(d_d));

    hipPeekAtLastError();
    hipDeviceSynchronize();
    return 0;
}